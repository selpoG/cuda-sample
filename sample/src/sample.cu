#include "hip/hip_runtime.h"
﻿#include <boost/version.hpp>
#include <iostream>
#include "sample.h"
#include "cuda_utility.h"

constexpr unsigned N = 15;

__global__ void hello(int a) { printf("hello from GPU! %d (%d\n", a, N); }

int Annealer::hoge(int a)
{
	CALL_KERNEL(hello, 3, 4)(a);
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	std::cout << "hello " << a << std::endl;
	std::cout << "boostバージョン:" << BOOST_VERSION << std::endl;
	std::cout << "boostライブラリバージョン:" << BOOST_LIB_VERSION << std::endl;
	return a + 1;
}
